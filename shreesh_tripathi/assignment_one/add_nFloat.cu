#include "hip/hip_runtime.h"
#include<iostream>
#include<math.h>
using namespace std;

//Global variable 
unsigned long long int size;

//Function: CPU
void cpu_add(float* h_a, float* h_b, float* h_d){
    for (int i = 0; i < size; i++)
    {
        h_d[i] = h_a[i] + h_b[i];
    }
}

//Kernel: GPU
__global__ void Add(float* d_a, float* d_b, float* d_c){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    d_c[id] = d_a[id] + d_b[id];
}


int main(){
    //Init GPU pointers
    float *d_a = NULL;
    float *d_b = NULL;
    float *d_c = NULL;

    cout << "Enter number of elements: "; cin >> size;
    //Init input arrays
    float h_a[size], h_b[size], h_c[size], h_d[size];
    
    //Log arrays
    cout << "\nEnter " << size << " numbers for array A " << endl;
    for(int i = 0; i < size; i++){
        cin >> h_a[i];
    }
    cout << "\nEnter " << size << " numbers for array B " << endl;
    for(int i = 0; i < size; i++){
        cin >> h_b[i];
    }

    //Display arrays
    cout << "\nArray A logged: " << endl;
    for(int i = 0; i < size; i++){
        cout << h_a[i] << "  ";
    }
    cout << "\n\nArray B logged: " << endl;
    for(int i = 0; i < size; i++){
        cout << h_b[i] << "  ";
    }

    // Computing using CPU
    cpu_add(h_a, h_b, h_d);

    //CPU result
    cout << "\n\nCPU Result" << endl;
    for (int i = 0; i < size; i++)
    {
        cout << h_d[i] << "  ";
    }

    //Computing using GPU    
    //Allocating memory in GPU pointed by d_x (x=a,b,c)
    int arr_bytes = size * sizeof(float);
    hipMalloc((void**)&d_a, arr_bytes);
    hipMalloc((void**)&d_b, arr_bytes);
    hipMalloc((void**)&d_c, arr_bytes);

    // Copying CPU -->  GPU memory
    hipMemcpy(d_a, h_a, arr_bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, arr_bytes, hipMemcpyHostToDevice);

    //Kernel call; Block: b, Threads: 1024 (max possible)
    //b = upper_ceil(1.0*size/1024)
    
    Add<<< ceil(1.0*size/1024), 1024 >>>(d_a, d_b, d_c);

    //Copying GPU --> CPU memory
    hipMemcpy(h_c, d_c, arr_bytes, hipMemcpyDeviceToHost);

    //GPU Result 
    cout << "\n\nGPU Result" << endl;
    for(int i = 0;i < size; i++){
        cout << h_c[i] << " ";
    }

    //De-allocating memory
    hipFree(d_a); hipFree(d_b); hipFree(d_c); 
}