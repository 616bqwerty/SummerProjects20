
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;

__global__ void Add(float *array1, float *array2, float *out,unsigned int *i)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x + *i;
	out[id] = array1[id] + array2[id];
	
}

int main()
{
	unsigned int array_Size, bigBlock = 1024*1024;
		
	cout << "Enter the size of Array: ";
	cin >> array_Size;

	float h_array1[array_Size], h_array2[array_Size], h_out[array_Size];
	
	int array_Bytes = array_Size * sizeof(float);

	for(int i=0;i<array_Size;i++)
		h_array1[i] = i;

	for(int i=1,j=0;j<array_Size;j++){
		h_array2[j] = i;
		
	}

	float *d_array1, *d_array2, *d_out;
	unsigned int *d_i;
	
	hipMalloc((void**)&d_array1, array_Bytes);
	hipMalloc((void**)&d_array2,array_Bytes);
	hipMalloc((void**)&d_out,array_Bytes);
	hipMalloc((void**)&d_i,sizeof(int));
	
	hipMemcpy(d_array1, h_array1, array_Bytes, hipMemcpyHostToDevice);
	hipMemcpy(d_array2, h_array2, array_Bytes, hipMemcpyHostToDevice);
	
	if(array_Size > bigBlock)
	for(unsigned int *i=0;*i>=array_Size + bigBlock;*i+=bigBlock)
	{
		hipMemcpy(d_i, i, sizeof(int), hipMemcpyHostToDevice);
		Add<<<1024,1024>>>(d_array1, d_array2, d_out, d_i);
	}
	else
	{
		hipMemcpy(d_i, 0, sizeof(int), hipMemcpyHostToDevice);
		Add<<<ceil(1.0*array_Size/1024),1024>>>(d_array1, d_array2, d_out, d_i);
	}
	hipError_t e = hipMemcpy(h_out, d_out, array_Bytes, hipMemcpyDeviceToHost);
	
	if(e!=hipSuccess)
        	cout <<"CUDA error copying to Host: " << hipGetErrorString(e) << endl;
	
	for(int i=0;i<array_Size;i++)
		cout<< i << ". " << h_out[i] << "\n";
	
	hipFree(d_array1);
	hipFree(d_array2);
	hipFree(d_out); 
	hipFree(d_i);

}