
#include <hip/hip_runtime.h>
#include<iostream>
using namespace std;
int Array_Size_x,Array_Size_y;

__global__ void Sum(float* d_in1,float* d_in2, float* d_out,int* d_array_size_x,int* d_array_size_y)
{
	int j = threadIdx.x + blockIdx.x * blockDim.x;
    int k = threadIdx.y + blockIdx.y * blockDim.y;
    
	int i = j + k * *d_array_size_y;
	
    if (j < *d_array_size_y && k < *d_array_size_x) 
       d_out[i] = d_in1[i] + d_in2[i];
}
int main()
{
    cout << "Enter the array size (row , col) : ";
    cin >> Array_Size_x >> Array_Size_y;
	
    int Array_Bytes = Array_Size_x * sizeof(float) * Array_Size_y;  
	
	float *h_in1, *h_in2, *h_out;

    h_in1 = (float*)malloc(Array_Bytes);
    h_in2 = (float*)malloc(Array_Bytes);
    h_out = (float*)malloc(Array_Bytes);
	
    for(int i=0; i<Array_Size_x; i++)
    {
		for(int j = 0; j < Array_Size_y; j++)
			{ 
			h_in1[i*Array_Size_y + j] = i + 0.1;
            h_in2[i*Array_Size_y + j] = i + 0.2; 
			}
    }
	
	/*
	for(int i=0; i<Array_Size_x; i++)
		{for(int j = 0; j < Array_Size_y; j++)
			cout << h_in1[i*Array_Size_y + j] << " ";
			cout << endl;
			}
	for(int i=0; i<Array_Size_x; i++)
		{for(int j = 0; j < Array_Size_y; j++)
			cout << h_in2[i*Array_Size_y + j] << " ";
			cout << endl;
			}
		*/	
			
    float *d_in1,*d_in2, *d_out;
	int *d_array_size_x,*d_array_size_y;
	
    hipMalloc((void**)&d_in1, Array_Bytes);
	hipMalloc((void**)&d_in2, Array_Bytes);
    hipMalloc((void**)&d_out, Array_Bytes);
	hipMalloc((void**)&d_array_size_x, sizeof(int));
	hipMalloc((void**)&d_array_size_y, sizeof(int));

    hipMemcpy(d_in1, h_in1, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_in2, h_in2, Array_Bytes, hipMemcpyHostToDevice);
    hipMemcpy(d_array_size_y, &Array_Size_y, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_array_size_x, &Array_Size_x, sizeof(int), hipMemcpyHostToDevice);
	
	 dim3 dimBlock(32, 32);
	 dim3 dimGrid((int)ceil(1.0*Array_Size_y/dimBlock.x),(int)ceil(1.0*Array_Size_x/dimBlock.y));
	 
    Sum<<<dimGrid, dimBlock>>>(d_in1, d_in2, d_out,d_array_size_x,d_array_size_y);
	
    hipMemcpy(h_out, d_out, Array_Bytes, hipMemcpyDeviceToHost);

	for(int i=0; i<Array_Size_x; i++)
		{for(int j = 0; j < Array_Size_y; j++)
			cout << h_out[i*Array_Size_y + j]<< " ";
			cout << endl;
			}
			
    hipFree(d_in1);
	hipFree(d_in2);
    hipFree(d_out);
	hipFree(d_array_size_x);
	hipFree(d_array_size_y);
}